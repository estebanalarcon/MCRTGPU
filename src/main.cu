#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "photon.cuh"
#include "opacities.cuh"
#include "dust.cuh"
#include "stars.cuh"
#include "grid.cuh"
#include "frequencies.cuh"
#include "emissivity.cuh"
#include "global_functions.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__global__ void kernelGetTaupath(Photon* d_photons){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (d_photons[tid].onGrid){
    float rn = hiprand_uniform(&d_photons[tid].state);
    d_photons[tid].taupathTotal = - log(1.0-rn);
    d_photons[tid].taupathGone = 0.0;
  }
}

void countPhotonsInGrid(bool* h_onGrid, int numPhotons){
  int count=0;
  for (int i=0 ; i<numPhotons ; i++){
    if (h_onGrid[i]){
      count++;
    }
  }
  printf("countPhotons = %d\n",count);
}

void executePhotons(SimulationParameters param, Photon* d_photons,
  FrequenciesData* d_freqData, Stars* d_stars, Grid* d_grid, DustDensity* d_dustDensity,
  DustOpacity* d_dustOpacity, EmissivityDatabase* d_emissivityDb,  DustTemperature* d_dustTemperature, SimulationParameters* d_params){
    inicializeInicialPhoton<<<param.numParallelPhotons/param.blockSize,param.blockSize>>>(d_photons, d_freqData, d_stars, d_grid, d_dustDensity, d_dustOpacity);
    //inicializeInicialPhoton<<<1,1>>>(d_photons, d_freqData, d_stars, d_grid, d_dustDensity, d_dustOpacity);
    gpuErrchk(hipDeviceSynchronize());
    launchPhotons<<<param.numParallelPhotons/param.blockSize,param.blockSize>>>(d_photons, d_freqData, d_stars,d_grid,d_dustDensity,d_dustOpacity, d_emissivityDb, d_dustTemperature,d_params);
    //launchPhotons<<<1,1>>>(d_photons, d_freqData, d_stars,d_grid,d_dustDensity,d_dustOpacity, d_emissivityDb, d_dustTemperature);
    gpuErrchk(hipDeviceSynchronize());
  }

void executePhotons2(int blockSize, int numParallelPhotons,Photon* d_photons,
  FrequenciesData* d_freqData, Stars* d_stars, Grid* d_grid, DustDensity* d_dustDensity,
  DustOpacity* d_dustOpacity, EmissivityDatabase* d_emissivityDb,  DustTemperature* d_dustTemperature,
  bool* h_onGrid, bool* d_onGrid){
    inicializeInicialPhoton<<<numParallelPhotons/blockSize,blockSize>>>(d_photons, d_freqData, d_stars, d_grid, d_dustDensity, d_dustOpacity);
    gpuErrchk(hipDeviceSynchronize());
    kernelWalkNextEvent<<<numParallelPhotons/blockSize,blockSize>>>(d_photons,d_stars,d_dustDensity,d_dustOpacity,d_grid,d_dustTemperature);
    gpuErrchk(hipDeviceSynchronize());
    getPhotonsOnGrid<<<numParallelPhotons/blockSize,blockSize>>>(d_photons, d_onGrid);
    gpuErrchk(hipDeviceSynchronize());
    while (arePhotonsOnGrid(d_onGrid, h_onGrid, numParallelPhotons)){
      //countPhotonsInGrid(h_onGrid, numParallelPhotons);
      //printf("arePhotonsOnGrid%d...\n",cont);
      kernelDoAbsorptionEvent<<<numParallelPhotons/blockSize,blockSize>>>(d_photons,d_grid,d_freqData,d_stars,d_dustDensity,d_dustOpacity,d_emissivityDb,d_dustTemperature);
      gpuErrchk(hipDeviceSynchronize());
      //kernelDoScatteringEvent<<<numParallelPhotons/blockSize,blockSize>>>(d_photons);
      //gpuErrchk(hipDeviceSynchronize());
      kernelGetTaupath<<<numParallelPhotons/blockSize,blockSize>>>(d_photons);
      gpuErrchk(hipDeviceSynchronize());
      kernelWalkNextEvent<<<numParallelPhotons/blockSize,blockSize>>>(d_photons,d_stars,d_dustDensity,d_dustOpacity,d_grid,d_dustTemperature);
      gpuErrchk(hipDeviceSynchronize());
      getPhotonsOnGrid<<<numParallelPhotons/blockSize,blockSize>>>(d_photons, d_onGrid);
      gpuErrchk(hipDeviceSynchronize());

  }
  //printf("hola\n");
}

int main(int argc, char **argv){
  //hipDeviceReset();
  int dev = 0;
  hipDeviceProp_t deviceProp;
  gpuErrchk(hipGetDeviceProperties(&deviceProp, dev));
  gpuErrchk(hipSetDevice(dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);


  SimulationParameters param = readInputParameters(argc, argv);
  checkSimulationParameters(param);
  float nTemp = 1000.0;
  float temp0 = 0.01;
  float temp1 = 100000.0;
  //int numPhotons = 1024*10000;
  //int blockSize = 32;
  //int maxParallelPhotons = 1024*50;
  int numStreams = param.numPhotons/param.numParallelPhotons;
  printf("Number of streams = %d\n",numStreams);

  //bool* d_onGrid;
  //hipMalloc((void**)&(d_onGrid), sizeof(bool)*param.maxParallelPhotons );

  //bool* h_onGrid =(bool*) malloc(sizeof(bool)*param.maxParallelPhotons);

  //read, process input data and transfer to device
  Grid* grid = setUpGrid();
  Grid* d_grid = gridTransferToDevice(grid);

  FrequenciesData* freqData = setUpFrequenciesData();
  FrequenciesData* d_freqData = frequenciesTransferToDevice(freqData);

  Stars* stars = setUpStars(freqData,grid, param.numPhotons);
  Stars* d_stars = starsTransferToDevice(stars, freqData->numFrequencies);

  DustDensity* dustDensity = setUpDustDensity(grid);
  DustDensity* d_dustDensity = dustDensityTransferToDevice(dustDensity);

  DustOpacity* dustOpacity = setUpDustOpacity(freqData);
  DustOpacity* d_dustOpacity = dustOpacityTransferToDevice(dustOpacity, freqData->numFrequencies);

  EmissivityDatabase* emissivityDb = setUpEmissivityDatabase(nTemp, temp0, temp1, dustOpacity, freqData);
  EmissivityDatabase* d_emissivityDb = emissivityDbTransferToDevice(emissivityDb, freqData->numFrequencies, dustDensity->numSpec);

  DustTemperature* dustTemperature = setUpDustTemperature(dustDensity, grid);
  DustTemperature* d_dustTemperature  = dustTemperatureTransferToDevice(dustTemperature, dustDensity);

  int numSpec = dustDensity->numSpec;
  int numFrequencies = freqData->numFrequencies;
  int nx = grid->nCoord[0];
  int ny = grid->nCoord[1];
  int nz = grid->nCoord[2];

  //free cpu memory
  deallocateCumulEner(dustTemperature, numSpec, ny,nz);
  deallocateEmissivityDatabase(emissivityDb, nTemp, numSpec);
  deallocateDustOpacities(dustOpacity);
  deallocateDensities(dustDensity);
  deallocateStars(stars);
  deallocateFrequenciesData(freqData);
  deallocateGrid(grid);

  SimulationParameters* d_params = parametersTransferToDevice(param);
  Photon* photons1 = allocatePhotons(param.numParallelPhotons, numSpec, numFrequencies);
  Photon* d_photons1 = photonsTransferToDevice(photons1, param.numParallelPhotons, numSpec, numFrequencies);
  deallocatePhotons(photons1, param.numParallelPhotons);

  printf("End transfers...\n");

  for (int i=0 ; i<numStreams ; i++){
    printf("Stream %d...\n",i);
    executePhotons(param, d_photons1,d_freqData, d_stars,d_grid,d_dustDensity,d_dustOpacity, d_emissivityDb, d_dustTemperature, d_params);
    //executePhotons2(blockSize, maxParallelPhotons,d_photons1,d_freqData, d_stars,d_grid,d_dustDensity,d_dustOpacity, d_emissivityDb, d_dustTemperature,h_onGrid,d_onGrid);
  }
  printf("Convert energy to temperature...\n");
  int totalPositions = dustTemperature->totalPositions;
  //printf("Total cells: %d\n",totalPositions);

  int numBlocks = totalPositions / param.blockSize;
  if (totalPositions > param.blockSize*numBlocks){
    numBlocks++;
  }
  //printf("numBlocks for temp=%d \n",numBlocks);
  convertEnergyToTemperature<<<numBlocks,param.blockSize>>>(d_dustTemperature,d_dustDensity,d_grid,d_emissivityDb);
  gpuErrchk(hipDeviceSynchronize());
  //printEner<<<1,1>>>(d_dustTemperature,d_dustDensity);
  //gpuErrchk(hipDeviceSynchronize());


  writeDustTemperature(dustTemperature, d_dustTemperature, numSpec,nz,ny,nx);
  //deallocateDustTemperature(dustTemperature, numSpec, ny,nz);*/
  printf("End simulation\n");
  hipDeviceReset();
}
