#include "hip/hip_runtime.h"
#include "photon.cuh"
#include "emissivity.cuh"
#include "grid.cuh"
#include "global_functions.cuh"

__device__ int signs[2] = {-1,1};

__global__ void kernelDoAbsorptionEvent(Photon* d_photons, Grid* grid, FrequenciesData* freqData, Stars* stars,
  DustDensity* dustDensity, DustOpacity* dustOpacity, EmissivityDatabase* emissivityDb, DustTemperature* dustTemperature){
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (d_photons[tid].onGrid){
      if (!d_photons[tid].isScattering){
        int ix = d_photons[tid].gridPosition[0];
        int iy = d_photons[tid].gridPosition[1];
        int iz = d_photons[tid].gridPosition[2];
        divideAbsorvedEnergy(&d_photons[tid],stars,dustDensity, dustOpacity);
        addTemperatureDecoupled(&d_photons[tid], dustDensity, grid, emissivityDb, dustTemperature);
        for (int i=0 ; i<dustDensity->numSpec ; i++){
          d_photons[tid].tempLocal[i] = dustTemperature->temperatures[i][iz][iy][ix];
        }
        d_photons[tid].iFrequency = pickRandomFreqDb(emissivityDb, &d_photons[tid], dustDensity->numSpec, freqData->numFrequencies, d_photons[tid].tempLocal, d_photons[tid].enerPart);
        getRandomDirectionSimple(&d_photons[tid], &d_photons[tid].state);
      }
      return;
    }
    return;
  }

__global__ void kernelDoScatteringEvent(Photon* d_photons){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (d_photons[tid].onGrid){
    if (d_photons[tid].isScattering){
      getRandomDirectionSimple(&d_photons[tid], &d_photons[tid].state);
    }
    return;
  }
  return;
}

__global__ void getPhotonsOnGrid(Photon* d_photons, bool* d_photonsOnGrid){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  d_photonsOnGrid[tid] = d_photons[tid].onGrid;
  //printf("photon%d grid=%d\n",tid,d_photonsOnGrid[tid]);
}

__host__ void transferOnGridArrayToHost(bool* d_photonsOnGrid, bool* h_photonsOnGrid, int numPhotons){
  hipMemcpy(h_photonsOnGrid, d_photonsOnGrid,sizeof(bool)*numPhotons,hipMemcpyDeviceToHost);
  return;
}

__host__ bool arePhotonsOnGrid(bool* d_onGrid, bool* h_onGrid, int numPhotons){
  //transferOnGridArrayToHost(d_onGrid,h_onGrid,numPhotons );
  hipMemcpy(h_onGrid, d_onGrid,sizeof(bool)*numPhotons,hipMemcpyDeviceToHost);
  for (int i=0 ; i<numPhotons ; i++){
    if (h_onGrid[i]){
      return true;
    }
  }
  return false;
}
/*
__global__ void convertEnergyToDecoupledTemperature(Grid* grid,Photon* photon, DustDensity* dustDensity, DustTemperature* dustTemperature, EmissivityDatabase* emissivityDb){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int* gridPosition = getIxyzFromIndex(grid, tid);
  int ix = photon->gridPosition[0];
  int iy = photon->gridPosition[1];
  int iz = photon->gridPosition[2];
  double ener;
  for (int i=0 ; i<dustDensity->numSpec ; i++){
    ener = dustTemperature->cumulEner[i][iz][iy][ix]/(dustDensity->densities[i][iz][iy][ix]*grid->cellVolumes);
    if (ener>0){
      dustTemperature->temperatures[i][iz][iy][ix] = computeDusttempEnergyBd(emissivityDb, ener, i);
    }else{
      dustTemperature->temperatures[i][iz][iy][ix] = 0;
    }
  }
}*/
__global__ void testId(){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  printf("tid:%d, int %d\n",tid,5);
}
__global__ void testId2(){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  printf("tid:%d, int %d\n",tid,2);
}

__device__ void divideAbsorvedEnergy(Photon* photon, Stars* stars,
  DustDensity* dustDensity, DustOpacity* dustOpacity){
    //printf("in divideAbsorvedEnergy\n");
    int ix = photon->gridPosition[0];
    int iy = photon->gridPosition[1];
    int iz = photon->gridPosition[2];
    double alphaA=0;
    if (dustDensity->numSpec == 1){
      photon->enerPart[0] = stars->energies[photon->iStar];
    }else{
      for (int i=0 ; i<dustDensity->numSpec ; i++){
        photon->enerPart[i] = dustDensity->densities[i][iz][iy][ix] * dustOpacity->kappaA[i][photon->iFrequency];
        alphaA += photon->enerPart[i];
      }
      for (int i=0 ; i<dustDensity->numSpec ; i++){
        photon->enerPart[i] = stars->energies[photon->iStar] * photon->enerPart[i] / alphaA;
      }
    }
  }

__device__ void addTemperatureDecoupled(Photon* photon, DustDensity* dustDensity,
  Grid* grid, EmissivityDatabase* emissivityDb, DustTemperature* dustTemperature){
    //printf("in addTemperatureDecoupled\n");

    int ix = photon->gridPosition[0];
    int iy = photon->gridPosition[1];
    int iz = photon->gridPosition[2];
    double cumen;
    for (int iSpec=0 ; iSpec < dustDensity->numSpec ; iSpec++){
      cumen = dustTemperature->cumulEner[iSpec][iz][iy][ix] / (dustDensity->densities[iSpec][iz][iy][ix]* grid->cellVolumes);
      dustTemperature->temperatures[iSpec][iz][iy][ix] = computeDusttempEnergyBd(emissivityDb, cumen, iSpec);
    }
  }

__device__ void doAbsorptionEvent(Photon* photon, Grid* grid, FrequenciesData* freqData, Stars* stars,
  DustDensity* dustDensity, DustOpacity* dustOpacity, EmissivityDatabase* emissivityDb, DustTemperature* dustTemperature){
    int ix = photon->gridPosition[0];
    int iy = photon->gridPosition[1];
    int iz = photon->gridPosition[2];
    divideAbsorvedEnergy(photon,stars,dustDensity, dustOpacity);
    addTemperatureDecoupled(photon, dustDensity, grid, emissivityDb, dustTemperature);
    for (int i=0 ; i<dustDensity->numSpec ; i++){
      photon->tempLocal[i] = dustTemperature->temperatures[i][iz][iy][ix];
    }
    photon->iFrequency = pickRandomFreqDb(emissivityDb, photon, dustDensity->numSpec, freqData->numFrequencies, photon->tempLocal, photon->enerPart);
    //photon->iFrequency = 55;
    //printf("newInu=%d\n",photon->iFrequency);
  }

__device__ double advanceToNextPositionTest(Photon* photon, Grid* grid, double cellWallsX[], double cellWallsY[], double cellWallsZ[]){
  //obtain orientations. It is 0 (left,down) or 1 (right, up)
  int ix = floor(photon->direction[0])+1.0;
  int iy = floor(photon->direction[1])+1.0;
  int iz = floor(photon->direction[2])+1.0;

  photon->orientations[0]=ix;
  photon->orientations[1]=iy;
  photon->orientations[2]=iz;

  //test shared memory
  //axis x
  photon->cellWalls[0] = cellWallsX[photon->gridPosition[0]+ix];
  //axis y
  photon->cellWalls[1] = cellWallsY[photon->gridPosition[1]+iy];
  //axis z
  photon->cellWalls[2] = cellWallsZ[photon->gridPosition[2]+iz];
  //printf("cellWallsX=%10.10lg\n",cellWallsX[photon->gridPosition[0]+ix]);


  //get 3 walls of grid position
  //getCellWalls(photon, grid, photon->gridPosition, photon->orientations);

  //distance to axis x
  photon->distances[0] = (photon->cellWalls[0] - photon->rayPosition[0]) / photon->direction[0];
  //distance to axis y
  photon->distances[1] = (photon->cellWalls[1] - photon->rayPosition[1]) / photon->direction[1];
  //distance to axis z
  photon->distances[2] = (photon->cellWalls[2] - photon->rayPosition[2]) / photon->direction[2];
  //printf("distances: %lf, %lf, %lf\n",distances[0],distances[1],distances[2]);

  //calculate min distance
  double tmp = fmin(photon->distances[0], photon->distances[1]);
  double minDistance = fmin(tmp, photon->distances[2]);
  //printf("minDistance: %lf\n", minDistance);

  //obtain minimun's axis
  //can be more than 1 (corners)
  int count = 0;
  int indexes[3] = {-1,-1,-1};
  for (int i=0 ; i<3 ; i++){
    if (photon->distances[i] == minDistance){
      indexes[count]=i;
      count++;
    }
  }
  //printf("minDistance, count: %lf, %d\n", minDistance, count);

  //update ray position
  photon->rayPosition[0] += minDistance*photon->direction[0];
  photon->rayPosition[1] += minDistance*photon->direction[1];
  photon->rayPosition[2] += minDistance*photon->direction[2];

  //avoid bug assign cellWall to ray position
  //update grid position with signs
  for (int i=0 ; i<count ; i++){
    photon->rayPosition[indexes[i]] = photon->cellWalls[indexes[i]];
    photon->gridPosition[indexes[i]] += signs[photon->orientations[indexes[i]]];
  }

  //is photon on the grid or outside?
  int nx = grid->nCoord[0];
  int ny = grid->nCoord[1];
  int nz = grid->nCoord[2];
  bool onX = (photon->gridPosition[0] >= 0) && (photon->gridPosition[0] < nx);
  bool onY = (photon->gridPosition[1] >= 0) && (photon->gridPosition[1] < ny);
  bool onZ = (photon->gridPosition[2] >= 0) && (photon->gridPosition[2] < nz);
  photon->onGrid = (onX && onY && onZ);
  return minDistance;

}

__device__ double advanceToNextPosition(Photon* photon, Grid* grid){
  //obtain orientations. It is 0 (left,down) or 1 (right, up)
  int ix = floor(photon->direction[0])+1.0;
  int iy = floor(photon->direction[1])+1.0;
  int iz = floor(photon->direction[2])+1.0;

  photon->orientations[0]=ix;
  photon->orientations[1]=iy;
  photon->orientations[2]=iz;


  //get 3 walls of grid position
  getCellWalls(photon, grid, photon->gridPosition, photon->orientations);

  //distance to axis x
  photon->distances[0] = (photon->cellWalls[0] - photon->rayPosition[0]) / photon->direction[0];
  //distance to axis y
  photon->distances[1] = (photon->cellWalls[1] - photon->rayPosition[1]) / photon->direction[1];
  //distance to axis z
  photon->distances[2] = (photon->cellWalls[2] - photon->rayPosition[2]) / photon->direction[2];
  //printf("distances: %lf, %lf, %lf\n",distances[0],distances[1],distances[2]);

  //calculate min distance
  double tmp = fmin(photon->distances[0], photon->distances[1]);
  double minDistance = fmin(tmp, photon->distances[2]);
  //printf("minDistance: %lf\n", minDistance);

  //obtain minimun's axis
  //can be more than 1 (corners)
  int count = 0;
  int indexes[3] = {-1,-1,-1};
  for (int i=0 ; i<3 ; i++){
    if (photon->distances[i] == minDistance){
      indexes[count]=i;
      count++;
    }
  }
  //printf("minDistance, count: %lf, %d\n", minDistance, count);

  //update ray position
  photon->rayPosition[0] += minDistance*photon->direction[0];
  photon->rayPosition[1] += minDistance*photon->direction[1];
  photon->rayPosition[2] += minDistance*photon->direction[2];

  //avoid bug assign cellWall to ray position
  //update grid position with signs
  for (int i=0 ; i<count ; i++){
    photon->rayPosition[indexes[i]] = photon->cellWalls[indexes[i]];
    photon->gridPosition[indexes[i]] += signs[photon->orientations[indexes[i]]];
  }

  //is photon on the grid or outside?
  photon->onGrid = photonIsOnGrid(photon, grid);
  return minDistance;
}

__device__ bool photonIsOnGrid(Photon* photon, Grid* grid){
  bool onX = (photon->gridPosition[0] >= 0) && (photon->gridPosition[0] < grid->nCoord[0]);
  bool onY = (photon->gridPosition[1] >= 0) && (photon->gridPosition[1] < grid->nCoord[1]);
  bool onZ = (photon->gridPosition[2] >= 0) && (photon->gridPosition[2] < grid->nCoord[2]);
  bool onGrid = (onX && onY && onZ);
  return onGrid;
}

__device__ void getTaupath(Photon* photon, hiprandState* state){
  float rn = hiprand_uniform(state);
  photon->taupathTotal = - log(1.0-rn);
  photon->taupathGone = 0.0;
}

__device__ void findNewFrequencyInu(Photon* photon, hiprandState* state, double* specCum, FrequenciesData* freqData){
  float rn = hiprand_uniform(state);
  int freq= (int)photon->iFrequency;
  huntDouble(specCum, freqData->numFrequencies+1, (double) rn, &freq);
  photon->iFrequency = (short)freq;
  //printf("rayInu = %d\n",rayInu);
  //return rayInu;
}

__device__ void checkUnitVector(double* direction){
    double module = sqrt(direction[0]*direction[0] + direction[1]*direction[1] + direction[2]*direction[2]);
    if (fabs(module - 1) > 1e-10){
      printf("Error unity vector\n");
    }else{
      printf("correct\n");
    }
}

__device__ void getRandomDirectionSimple(Photon* photon, hiprandState *state){
  //unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  float dirx, diry, dirz;
  bool equalZero = true;
  bool equalOne = true;
  float tmp, module;
  while (equalZero || equalOne){
    dirx = 2*hiprand_uniform(state)-1;
    diry = 2*hiprand_uniform(state)-1;
    dirz = 2*hiprand_uniform(state)-1;
    tmp = dirx*dirx + diry*diry + dirz*dirz;
    module = 1.0/sqrt(tmp);
    dirx = dirx*module;
    diry = diry*module;
    dirz = dirz*module;
    equalZero = (dirx==0.0) || (diry==0.0)|| (dirz==0.0);
    equalOne =  (dirx==1.0) || (diry==1.0)|| (dirz==1.0);
    //printf("direction %d=%lf,%lf,%lf\n",tid,photon->direction[0],photon->direction[1],photon->direction[2]);
  }

  photon->direction[0] = dirx;
  photon->direction[1] = diry;
  photon->direction[2] = dirz;

  /*printf("direction.x %lg\n",dirx);
  printf("direction.y %lg\n",diry);
  printf("direction.z %lg\n",dirz);*/
  return;
}
__device__ void findStar(Photon* photon, Stars* d_stars, hiprandState* state){
  photon->iStar = 0;
  if (d_stars->numStars > 1){
    float rn = hiprand_uniform(state);
    int istar=(int)photon->iStar;
    huntDouble(d_stars->luminositiesCum, d_stars->numStars+1, (double) rn, &istar);
    photon->iStar = (short) istar;
  }
}

__host__ void setUpPhoton(Photon* photon, int numSpec, int numFreq){

  photon->alphaASpec = (double*)malloc(sizeof(double)*numSpec);
  photon->alphaSSpec = (double*)malloc(sizeof(double)*numSpec);
  photon->dbCumul = (float*)malloc(sizeof(float)*(numFreq+1));
  photon->enerCum = (double*)malloc(sizeof(double)*(numSpec+1));
  photon->enerPart = (double*)malloc(sizeof(double)*numSpec);
  photon->tempLocal = (double*)malloc(sizeof(double)*numSpec);
  photon->onGrid = true;
  /*photon->taupathGone = 0;
  photon->taupathTotal = 0;
  photon->iFrequency = 0;
  for (int i=0 ; i<numSpec ; i++){
    photon->alphaASpec[i] = 0;
    photon->alphaSSpec[i] = 0;
  }*/
}

__host__ void freePhoton(Photon* photon){
  free(photon->alphaASpec);
  free(photon->alphaSSpec);
  free(photon->dbCumul);
  free(photon->enerCum);
  free(photon->enerPart);
  free(photon->tempLocal);
}

__device__ void calculateOpacityCoefficients(double minorDistance, Photon* photon, DustDensity* dustDensity,
  DustOpacity* dustOpacity){
    int ix = photon->prevGridPosition[0];
    int iy = photon->prevGridPosition[1];
    int iz = photon->prevGridPosition[2];
    photon->opacCoeff.alphaATotal=0;
    photon->opacCoeff.alphaSTotal=0;
    for (int iSpec=0 ; iSpec<dustDensity->numSpec ; iSpec++){
      //printf("densities: %10.10lg\n",dustDensity->densities[iSpec][iz][iy][ix]);
      //printf("kappaA: %10.10lg\n",dustOpacity->kappaA[iSpec][photon->iFrequency]);
      photon->alphaASpec[iSpec] = dustDensity->densities[iSpec][iz][iy][ix]*dustOpacity->kappaA[iSpec][photon->iFrequency];
      photon->alphaSSpec[iSpec] = dustDensity->densities[iSpec][iz][iy][ix]*dustOpacity->kappaS[iSpec][photon->iFrequency];
      photon->opacCoeff.alphaATotal += photon->alphaASpec[iSpec];
      photon->opacCoeff.alphaSTotal += photon->alphaSSpec[iSpec];
    }
    photon->opacCoeff.alphaTotal = photon->opacCoeff.alphaATotal+photon->opacCoeff.alphaSTotal;
    photon->opacCoeff.albedo = photon->opacCoeff.alphaSTotal/photon->opacCoeff.alphaTotal;
    photon->opacCoeff.dtau = photon->opacCoeff.alphaTotal * minorDistance;
    //printf("dtau: %10.10lg\n",opacCoeff->dtau);
}

__device__ void inicializePositionPhoton(Photon* photon, Stars* d_stars,
  Grid* d_grid){
  //rayPosition
  photon->rayPosition[0] = d_stars->positions[photon->iStar][0];
  photon->rayPosition[1] = d_stars->positions[photon->iStar][1];
  photon->rayPosition[2] = d_stars->positions[photon->iStar][2];

  //gridPosition
  convertRayToGrid(photon, d_grid);
}

__device__ void walkNextEvent(Photon* d_photons, Stars* d_stars, DustDensity* d_dustDensity, DustOpacity* d_dustOpacity,
    Grid* d_grid, DustTemperature* d_dustTemperature, double cellWallsX[], double cellWallsY[], double cellWallsZ[]){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  double minorDistance, fraction, dum, addTmp;
  bool carryOn = true;
  //printf("cellWallsX 0 = %10.10lg\n",cellWallsX[0] );
  //printf("cellWallsX 1 = %10.10lg\n",cellWallsX[1] );
  while (carryOn){

    d_photons[tid].prevRayPosition[0] = d_photons[tid].rayPosition[0];
    d_photons[tid].prevRayPosition[1] = d_photons[tid].rayPosition[1];
    d_photons[tid].prevRayPosition[2] = d_photons[tid].rayPosition[2];

    d_photons[tid].prevGridPosition[0] = d_photons[tid].gridPosition[0];
    d_photons[tid].prevGridPosition[1] = d_photons[tid].gridPosition[1];
    d_photons[tid].prevGridPosition[2] = d_photons[tid].gridPosition[2];
    //printf("start advanceToNextPosition\n");
    minorDistance = advanceToNextPositionTest(&d_photons[tid], d_grid,cellWallsX,cellWallsY,cellWallsZ);
    //minorDistance = advanceToNextPosition(&d_photons[tid], d_grid);
    //if (d_photons[tid].gridPosition[0]==0 && d_photons[tid].gridPosition[1]==0 && d_photons[tid].gridPosition[2]==0){
    //  printf("GridPosition: %d, %d, %d, onGrid = %d\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2], d_photons[tid].onGrid);
    //}

    calculateOpacityCoefficients(minorDistance, &d_photons[tid], d_dustDensity, d_dustOpacity);

    //printf("alphaSTotal=%10.10lg\n", opacCoeff->alphaSTotal);

    if (d_photons[tid].taupathGone + d_photons[tid].opacCoeff.dtau > d_photons[tid].taupathTotal){
      //printf("taupathGone=%lf, dtau=%lf, taupathTotal=%lf\n", d_photons[tid].taupathGone,d_photons[tid].opacCoeff.dtau,d_photons[tid].taupathTotal);
      //printf("\nReached end point!\n\n");
      fraction = (d_photons[tid].taupathTotal - d_photons[tid].taupathGone)/d_photons[tid].opacCoeff.dtau;
      //printf("fr=%lf\n", fraction)
      //update ray position

      d_photons[tid].rayPosition[0] = d_photons[tid].prevRayPosition[0] + fraction * (d_photons[tid].rayPosition[0] - d_photons[tid].prevRayPosition[0]);
      d_photons[tid].rayPosition[1] = d_photons[tid].prevRayPosition[1] + fraction * (d_photons[tid].rayPosition[1] - d_photons[tid].prevRayPosition[1]);
      d_photons[tid].rayPosition[2] = d_photons[tid].prevRayPosition[2] + fraction * (d_photons[tid].rayPosition[2] - d_photons[tid].prevRayPosition[2]);
      //update grid position
      d_photons[tid].gridPosition[0] = d_photons[tid].prevGridPosition[0];
      d_photons[tid].gridPosition[1] = d_photons[tid].prevGridPosition[1];
      d_photons[tid].gridPosition[2] = d_photons[tid].prevGridPosition[2];
      //index = getIndexFromIxyz(d_grid, photon->gridPosition);
      dum = (1.0-d_photons[tid].opacCoeff.albedo) * (d_photons[tid].taupathTotal-d_photons[tid].taupathGone) * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
      //printf("dum=%10.10lg\n",dum);
      for (int i=0 ; i<d_dustDensity->numSpec ; i++){
        addTmp = dum*d_photons[tid].alphaASpec[i];
        //d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]] += addTmp;

        atomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]), addTmp);
        if (d_photons[tid].gridPosition[0]==0 && d_photons[tid].gridPosition[1]==0 && d_photons[tid].gridPosition[2]==0){
          printf("cumulEner %d, %d, %d = %10.10lg\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2],
          d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
        }
        //d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]] += addTmp;
        //printf("cumulEner=%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]);
      }
      carryOn = false;
    }else{
      dum = (1.0-d_photons[tid].opacCoeff.albedo) * d_photons[tid].opacCoeff.dtau * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
      for (int i=0 ; i<d_dustDensity->numSpec ; i++){
        addTmp = dum*d_photons[tid].alphaASpec[i];
        //d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]] += addTmp;
        atomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]), addTmp);
        //if (d_photons[tid].prevGridPosition[0]==0 && d_photons[tid].prevGridPosition[1]==0 && d_photons[tid].prevGridPosition[2]==0){
        //  printf("cumulEner %d, %d, %d = %10.10lg\n",d_photons[tid].prevGridPosition[0],d_photons[tid].prevGridPosition[1],d_photons[tid].prevGridPosition[2],
        //  d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
        //}
        //d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]] += addTmp;
        //printf("cumulEner =%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
      }

      d_photons[tid].taupathGone += d_photons[tid].opacCoeff.dtau;
      carryOn = d_photons[tid].onGrid;
    }
  }
  //printf("FinalRayPosition: %lf, %lf, %lf\n",d_photons[tid].rayPosition[0],d_photons[tid].rayPosition[1],d_photons[tid].rayPosition[2]);
  //printf("FinalGridPosition: %d, %d, %d\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2]);
  float rn = hiprand_uniform(&d_photons[tid].state);
  //printf("rn=%lf\n",rn);
  d_photons[tid].isScattering = rn < d_photons[tid].opacCoeff.albedo;
  //printf("end walkEvent\n");
}
/*
__device__ void walkNextEvent(Photon* d_photons, Stars* d_stars, DustDensity* d_dustDensity, DustOpacity* d_dustOpacity,
    Grid* d_grid, DustTemperature* d_dustTemperature){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  double minorDistance, fraction, dum, addTmp;
  bool carryOn = true;
  //printf("cellWallsX 0 = %10.10lg\n",cellWallsX[0] );
  //printf("cellWallsX 1 = %10.10lg\n",cellWallsX[1] );
  while (carryOn){

    d_photons[tid].prevRayPosition[0] = d_photons[tid].rayPosition[0];
    d_photons[tid].prevRayPosition[1] = d_photons[tid].rayPosition[1];
    d_photons[tid].prevRayPosition[2] = d_photons[tid].rayPosition[2];

    d_photons[tid].prevGridPosition[0] = d_photons[tid].gridPosition[0];
    d_photons[tid].prevGridPosition[1] = d_photons[tid].gridPosition[1];
    d_photons[tid].prevGridPosition[2] = d_photons[tid].gridPosition[2];
    //printf("start advanceToNextPosition\n");
    //minorDistance = advanceToNextPositionTest(&d_photons[tid], d_grid,cellWallsX,cellWallsY,cellWallsZ);
    minorDistance = advanceToNextPosition(&d_photons[tid], d_grid);
    if (d_photons[tid].gridPosition[0]==0 && d_photons[tid].gridPosition[1]==0 && d_photons[tid].gridPosition[2]==0){
      printf("GridPosition: %d, %d, %d, onGrid = %d\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2], d_photons[tid].onGrid);
    }

    if (d_photons[tid].onGrid){
      calculateOpacityCoefficients(minorDistance, &d_photons[tid], d_dustDensity, d_dustOpacity);

      //printf("alphaSTotal=%10.10lg\n", opacCoeff->alphaSTotal);

      if (d_photons[tid].taupathGone + d_photons[tid].opacCoeff.dtau > d_photons[tid].taupathTotal){
        //printf("taupathGone=%lf, dtau=%lf, taupathTotal=%lf\n", d_photons[tid].taupathGone,d_photons[tid].opacCoeff.dtau,d_photons[tid].taupathTotal);
        //printf("\nReached end point!\n\n");
        fraction = (d_photons[tid].taupathTotal - d_photons[tid].taupathGone)/d_photons[tid].opacCoeff.dtau;
        //printf("fr=%lf\n", fraction)
        //update ray position

        d_photons[tid].rayPosition[0] = d_photons[tid].prevRayPosition[0] + fraction * (d_photons[tid].rayPosition[0] - d_photons[tid].prevRayPosition[0]);
        d_photons[tid].rayPosition[1] = d_photons[tid].prevRayPosition[1] + fraction * (d_photons[tid].rayPosition[1] - d_photons[tid].prevRayPosition[1]);
        d_photons[tid].rayPosition[2] = d_photons[tid].prevRayPosition[2] + fraction * (d_photons[tid].rayPosition[2] - d_photons[tid].prevRayPosition[2]);
        //update grid position
        d_photons[tid].gridPosition[0] = d_photons[tid].prevGridPosition[0];
        d_photons[tid].gridPosition[1] = d_photons[tid].prevGridPosition[1];
        d_photons[tid].gridPosition[2] = d_photons[tid].prevGridPosition[2];
        //index = getIndexFromIxyz(d_grid, photon->gridPosition);
        dum = (1.0-d_photons[tid].opacCoeff.albedo) * (d_photons[tid].taupathTotal-d_photons[tid].taupathGone) * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
        //printf("dum=%10.10lg\n",dum);
        for (int i=0 ; i<d_dustDensity->numSpec ; i++){
          addTmp = dum*d_photons[tid].alphaASpec[i];
          //d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]] += addTmp;

          atomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]), addTmp);
          if (d_photons[tid].gridPosition[0]==0 && d_photons[tid].gridPosition[1]==0 && d_photons[tid].gridPosition[2]==0){
            printf("cumulEner %d, %d, %d = %10.10lg\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2],
            d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
          }
          //d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]] += addTmp;
          //printf("cumulEner=%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]);
        }
        carryOn = false;
      }else{
        dum = (1.0-d_photons[tid].opacCoeff.albedo) * d_photons[tid].opacCoeff.dtau * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
        for (int i=0 ; i<d_dustDensity->numSpec ; i++){
          addTmp = dum*d_photons[tid].alphaASpec[i];
          //d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]] += addTmp;
          atomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]), addTmp);
          if (d_photons[tid].prevGridPosition[0]==0 && d_photons[tid].prevGridPosition[1]==0 && d_photons[tid].prevGridPosition[2]==0){
            printf("cumulEner %d, %d, %d = %10.10lg\n",d_photons[tid].prevGridPosition[0],d_photons[tid].prevGridPosition[1],d_photons[tid].prevGridPosition[2],
            d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
          }
          //d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]] += addTmp;
          //printf("cumulEner =%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
        }

        d_photons[tid].taupathGone += d_photons[tid].opacCoeff.dtau;
      }
      //if photon is outside of grid
    }else{
      carryOn = false;
    }
          //printf("Ongrid? = %d\n",d_photons[tid].onGrid);
  }
  //printf("FinalRayPosition: %lf, %lf, %lf\n",d_photons[tid].rayPosition[0],d_photons[tid].rayPosition[1],d_photons[tid].rayPosition[2]);
  //printf("FinalGridPosition: %d, %d, %d\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2]);
  float rn = hiprand_uniform(&d_photons[tid].state);
  //printf("rn=%lf\n",rn);
  d_photons[tid].isScattering = rn < d_photons[tid].opacCoeff.albedo;
  //printf("end walkEvent\n");
}*/

__global__ void kernelWalkNextEvent(Photon* d_photons, Stars* d_stars, DustDensity* d_dustDensity, DustOpacity* d_dustOpacity,
  Grid* d_grid, DustTemperature* d_dustTemperature){
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (d_photons[tid].onGrid){
      //printf("startGridPosition %d: %d, %d, %d, ongrid=%d\n",tid,d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2],d_photons[tid].onGrid);

      double minorDistance, fraction, dum, addTmp;
      bool carryOn = true;

      while (carryOn){
        //printf("\n");
        d_photons[tid].prevRayPosition[0] = d_photons[tid].rayPosition[0];
        d_photons[tid].prevRayPosition[1] = d_photons[tid].rayPosition[1];
        d_photons[tid].prevRayPosition[2] = d_photons[tid].rayPosition[2];

        d_photons[tid].prevGridPosition[0] = d_photons[tid].gridPosition[0];
        d_photons[tid].prevGridPosition[1] = d_photons[tid].gridPosition[1];
        d_photons[tid].prevGridPosition[2] = d_photons[tid].gridPosition[2];

        minorDistance = advanceToNextPosition(&d_photons[tid], d_grid);
        if (d_photons[tid].onGrid){
          calculateOpacityCoefficients(minorDistance, &d_photons[tid], d_dustDensity, d_dustOpacity);

          //printf("alphaSTotal=%10.10lg\n", opacCoeff->alphaSTotal);

          if (d_photons[tid].taupathGone + d_photons[tid].opacCoeff.dtau > d_photons[tid].taupathTotal){
            //printf("taupathGone=%lf, dtau=%lf, taupathTotal=%lf\n", d_photons[tid].taupathGone,d_photons[tid].opacCoeff.dtau,d_photons[tid].taupathTotal);
            //printf("\nReached end point!\n\n");
            fraction = (d_photons[tid].taupathTotal - d_photons[tid].taupathGone)/d_photons[tid].opacCoeff.dtau;
            //printf("fr=%lf\n", fraction)
            //update ray position

            d_photons[tid].rayPosition[0] = d_photons[tid].prevRayPosition[0] + fraction * (d_photons[tid].rayPosition[0] - d_photons[tid].prevRayPosition[0]);
            d_photons[tid].rayPosition[1] = d_photons[tid].prevRayPosition[1] + fraction * (d_photons[tid].rayPosition[1] - d_photons[tid].prevRayPosition[1]);
            d_photons[tid].rayPosition[2] = d_photons[tid].prevRayPosition[2] + fraction * (d_photons[tid].rayPosition[2] - d_photons[tid].prevRayPosition[2]);
            //update grid position
            d_photons[tid].gridPosition[0] = d_photons[tid].prevGridPosition[0];
            d_photons[tid].gridPosition[1] = d_photons[tid].prevGridPosition[1];
            d_photons[tid].gridPosition[2] = d_photons[tid].prevGridPosition[2];
            //index = getIndexFromIxyz(d_grid, photon->gridPosition);
            dum = (1-d_photons[tid].opacCoeff.albedo) * (d_photons[tid].taupathTotal-d_photons[tid].taupathGone) * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
            //printf("dum=%10.10lg\n",dum);
            for (int i=0 ; i<d_dustDensity->numSpec ; i++){
              addTmp = dum*d_photons[tid].alphaASpec[i];
              doubleAtomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]), addTmp);
              //printf("cumulEner=%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]);
            }
            carryOn = false;
          }else{
            //index = getIndexFromIxyz(d_grid, photon->prevGridPosition);
            //printf("dum=%10.10lg\n",dum);
            dum = (1-d_photons[tid].opacCoeff.albedo) * d_photons[tid].opacCoeff.dtau * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
            for (int i=0 ; i<d_dustDensity->numSpec ; i++){
              addTmp = dum*d_photons[tid].alphaASpec[i];
              doubleAtomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]), addTmp);
              //printf("cumulEner =%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
            }
            d_photons[tid].taupathGone += d_photons[tid].opacCoeff.dtau;
          }
          //if photon is outside of grid
        }else{
          carryOn = false;
        }
              //printf("Ongrid? = %d\n",d_photons[tid].onGrid);
      }
      //printf("FinalGridPosition %d: %d, %d, %d, ongrid=%d\n",tid,d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2],d_photons[tid].onGrid);
      //printf("FinalRayPosition %d: %lf, %lf, %lf\n",tid, d_photons[tid].rayPosition[0],d_photons[tid].rayPosition[1],d_photons[tid].rayPosition[2]);
      float rn = hiprand_uniform(&d_photons[tid].state);
      //printf("rn=%lf\n",rn);
      d_photons[tid].isScattering = rn < d_photons[tid].opacCoeff.albedo;
    }
}

__global__ void inicializeInicialPhoton(Photon* d_photons, FrequenciesData* d_freqData, Stars* d_stars, Grid* d_grid,
  DustDensity* d_dustDensity, DustOpacity* d_dustOpacity){
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    //hiprandState state;
    hiprand_init((unsigned long long)clock()*tid, tid, 0, &d_photons[tid].state);
    //photons[tid] = setUpPhoton(d_dustDensity->numSpec, d_freqData->numFrequencies);
    findStar(&d_photons[tid],d_stars,&d_photons[tid].state);
    inicializePositionPhoton(&d_photons[tid], d_stars, d_grid);
    getRandomDirectionSimple(&d_photons[tid], &d_photons[tid].state);
    findNewFrequencyInu(&d_photons[tid],&d_photons[tid].state, d_stars->specCum[d_photons[tid].iStar], d_freqData);
    getTaupath(&d_photons[tid], &d_photons[tid].state);
    d_photons[tid].onGrid = true;
    //printf("iFrequency%d, iStar%d, taupath%lf, direction:%lf,%lf,%lf\n",
    //d_photons[tid].iFrequency, d_photons[tid].iStar, d_photons[tid].taupathTotal,
    //d_photons[tid].direction[0],d_photons[tid].direction[1],d_photons[tid].direction[2]);
}
/*
__global__ void restartValuesPhoton(Photon* d_photons){

}*/


__host__ Photon* allocatePhotons(int numPhotons, int numSpec, int numFreq){
  Photon* photons = (Photon*)malloc(sizeof(Photon)*numPhotons);
  for (int i=0 ; i< numPhotons ; i++){
    setUpPhoton(&photons[i], numSpec, numFreq);
  }
  return photons;
}

__host__ void deallocatePhotons(Photon* photons, int numPhotons){
  for (int i=0 ; i< numPhotons ; i++){
    freePhoton(&photons[i]);
  }
  free(photons);
}

__host__ Photon* photonsTransferToDevice(Photon* h_photons, int numPhotons, int numSpec, int numFreq){
  printf("Transfer photons to device...\n");
  Photon* d_photons;
  hipMalloc((void**)&(d_photons), sizeof(Photon)*numPhotons );
  Photon* photons = (Photon*)malloc(sizeof(Photon)*numPhotons);
  for (int i=0 ; i<numPhotons ; i++){
    hipMalloc((void**) &photons[i].alphaASpec, sizeof(double)*numSpec);
    hipMalloc((void**) &photons[i].alphaSSpec, sizeof(double)*numSpec);
    hipMalloc((void**) &photons[i].enerPart, sizeof(double)*numSpec);
    hipMalloc((void**) &photons[i].tempLocal, sizeof(double)*numSpec);
    hipMalloc((void**) &photons[i].enerCum, sizeof(double)*(numSpec+1));
    hipMalloc((void**) &photons[i].dbCumul, sizeof(float)*(numFreq+1));
  }
  hipMemcpy(d_photons, photons,sizeof(Photon)*numPhotons,hipMemcpyHostToDevice);
  free(photons);
  return d_photons;
}
/*
__global__ void printfPhotons(Photon* d_photons){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  d_photons[tid].opacCoeff.dtau = tid;
  printf("d_photons[tid].opacCoeff.dtau %lf\n",d_photons[tid].opacCoeff.dtau);
  //printf("Hola\n");
}*/

__global__ void launchPhotons(Photon* d_photons, FrequenciesData* d_freqData, Stars* d_stars, Grid* d_grid,
  DustDensity* d_dustDensity, DustOpacity* d_dustOpacity,
  EmissivityDatabase* d_emissivityDb, DustTemperature* d_dustTemperature){
  __shared__ double cellWallsX[257];
  __shared__ double cellWallsY[257];
  __shared__ double cellWallsZ[257];

  int scatteringMode = 1;
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

  if (threadIdx.x < 63){
    for (int i=0 ; i<4 ; i++){
      int index = threadIdx.x*4+i;
      cellWallsX[index] = d_grid->cellWallsX[index];
      cellWallsY[index] = d_grid->cellWallsY[index];
      cellWallsZ[index] = d_grid->cellWallsZ[index];
      //printf("cellWallsX %d = %10.10lg\n",index,cellWallsX[index] );
    }
  }
  if (threadIdx.x == 63){
    for (int i=0 ; i<6 ; i++){
      int index = threadIdx.x*4+i;
      cellWallsX[index] = d_grid->cellWallsX[index];
      cellWallsY[index] = d_grid->cellWallsY[index];
      cellWallsZ[index] = d_grid->cellWallsZ[index];
    }
  }
  __syncthreads();
  //printf("hola\n");
  walkNextEvent(d_photons, d_stars, d_dustDensity, d_dustOpacity, d_grid, d_dustTemperature, cellWallsX, cellWallsY, cellWallsZ);
  //walkNextEvent(d_photons, d_stars, d_dustDensity, d_dustOpacity, d_grid, d_dustTemperature);
  while(d_photons[tid].onGrid){

    if (d_photons[tid].isScattering){
      //Do Scattering event
      if (scatteringMode == 1){
        //random direction
        //printf("scaterring\n");
        getRandomDirectionSimple(&d_photons[tid], &d_photons[tid].state);
      }else if (scatteringMode == 2){
        //do henyey_greenstein_direction
      }

    }else{
      //printf("hola2\n");

      doAbsorptionEvent(&d_photons[tid], d_grid, d_freqData, d_stars, d_dustDensity,d_dustOpacity, d_emissivityDb, d_dustTemperature);
      //printf("newFreq = %d\n",d_photons[tid].iFrequency);
      //printf("start getRandomDirectionSimple\n");
      getRandomDirectionSimple(&d_photons[tid], &d_photons[tid].state);
      //printf("end getRandomDirectionSimple\n");
    }
    //printf("start getTaupath\n");
    getTaupath(&d_photons[tid], &d_photons[tid].state);
    //printf("start walkNextEvent, onGrid=%d\n",d_photons[tid].onGrid);
    walkNextEvent(d_photons, d_stars, d_dustDensity, d_dustOpacity, d_grid, d_dustTemperature,cellWallsX, cellWallsY, cellWallsZ);
    //walkNextEvent(d_photons, d_stars, d_dustDensity, d_dustOpacity, d_grid, d_dustTemperature);
    //printf("onGrid? %d\n",d_photons[tid].onGrid);
  }
}
