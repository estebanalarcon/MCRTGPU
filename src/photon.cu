#include "hip/hip_runtime.h"
#include "photon.cuh"
#include "emissivity.cuh"
#include "grid.cuh"
#include "global_functions.cuh"
#include "constants.cuh"

__device__ int signs[2] = {-1,1};
__device__ float precision[3] = {0,0.000001,-0.000001};
__device__ int valuesOrientations[3] = {0,1,1};

__global__ void kernelDoAbsorptionEvent(Photon* d_photons, Grid* grid, FrequenciesData* freqData, Stars* stars,
  DustDensity* dustDensity, DustOpacity* dustOpacity, EmissivityDatabase* emissivityDb, DustTemperature* dustTemperature){
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (d_photons[tid].onGrid){
      if (!d_photons[tid].isScattering){
        int ix = d_photons[tid].gridPosition[0];
        int iy = d_photons[tid].gridPosition[1];
        int iz = d_photons[tid].gridPosition[2];
        divideAbsorvedEnergy(&d_photons[tid],stars,dustDensity, dustOpacity);
        addTemperatureDecoupled(&d_photons[tid], dustDensity, grid, emissivityDb, dustTemperature);
        for (int i=0 ; i<dustDensity->numSpec ; i++){
          d_photons[tid].tempLocal[i] = dustTemperature->temperatures[i][iz][iy][ix];
        }
        d_photons[tid].iFrequency = pickRandomFreqDb(emissivityDb, &d_photons[tid], dustDensity->numSpec, freqData->numFrequencies, d_photons[tid].tempLocal, d_photons[tid].enerPart);
        getRandomDirectionSimple(&d_photons[tid], &d_photons[tid].state);
      }
      return;
    }
    return;
  }

__global__ void kernelDoScatteringEvent(Photon* d_photons){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (d_photons[tid].onGrid){
    if (d_photons[tid].isScattering){
      getRandomDirectionSimple(&d_photons[tid], &d_photons[tid].state);
    }
    return;
  }
  return;
}

__global__ void getPhotonsOnGrid(Photon* d_photons, bool* d_photonsOnGrid){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  d_photonsOnGrid[tid] = d_photons[tid].onGrid;
  //printf("photon%d grid=%d\n",tid,d_photonsOnGrid[tid]);
}

__host__ void transferOnGridArrayToHost(bool* d_photonsOnGrid, bool* h_photonsOnGrid, int numPhotons){
  hipMemcpy(h_photonsOnGrid, d_photonsOnGrid,sizeof(bool)*numPhotons,hipMemcpyDeviceToHost);
  return;
}

__host__ bool arePhotonsOnGrid(bool* d_onGrid, bool* h_onGrid, int numPhotons){
  //transferOnGridArrayToHost(d_onGrid,h_onGrid,numPhotons );
  hipMemcpy(h_onGrid, d_onGrid,sizeof(bool)*numPhotons,hipMemcpyDeviceToHost);
  for (int i=0 ; i<numPhotons ; i++){
    if (h_onGrid[i]){
      return true;
    }
  }
  return false;
}
/*
__global__ void convertEnergyToDecoupledTemperature(Grid* grid,Photon* photon, DustDensity* dustDensity, DustTemperature* dustTemperature, EmissivityDatabase* emissivityDb){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int* gridPosition = getIxyzFromIndex(grid, tid);
  int ix = photon->gridPosition[0];
  int iy = photon->gridPosition[1];
  int iz = photon->gridPosition[2];
  double ener;
  for (int i=0 ; i<dustDensity->numSpec ; i++){
    ener = dustTemperature->cumulEner[i][iz][iy][ix]/(dustDensity->densities[i][iz][iy][ix]*grid->cellVolumes);
    if (ener>0){
      dustTemperature->temperatures[i][iz][iy][ix] = computeDusttempEnergyBd(emissivityDb, ener, i);
    }else{
      dustTemperature->temperatures[i][iz][iy][ix] = 0;
    }
  }
}*/
__global__ void testId(){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  printf("tid:%d, int %d\n",tid,5);
}
__global__ void testId2(){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  printf("tid:%d, int %d\n",tid,2);
}

__device__ void divideAbsorvedEnergy(Photon* photon, Stars* stars,
  DustDensity* dustDensity, DustOpacity* dustOpacity){
    //printf("in divideAbsorvedEnergy\n");
    int ix = photon->gridPosition[0];
    int iy = photon->gridPosition[1];
    int iz = photon->gridPosition[2];
    double alphaA=0;
    if (dustDensity->numSpec == 1){
      photon->enerPart[0] = stars->energies[photon->iStar];
    }else{
      for (int i=0 ; i<dustDensity->numSpec ; i++){
        photon->enerPart[i] = dustDensity->densities[i][iz][iy][ix] * dustOpacity->kappaA[i][photon->iFrequency];
        alphaA += photon->enerPart[i];
      }
      for (int i=0 ; i<dustDensity->numSpec ; i++){
        photon->enerPart[i] = stars->energies[photon->iStar] * photon->enerPart[i] / alphaA;
      }
    }
  }

__device__ void addTemperatureDecoupled(Photon* photon, DustDensity* dustDensity,
  Grid* grid, EmissivityDatabase* emissivityDb, DustTemperature* dustTemperature){
    //printf("in addTemperatureDecoupled\n");

    int ix = photon->gridPosition[0];
    int iy = photon->gridPosition[1];
    int iz = photon->gridPosition[2];
    double cumen;
    for (int iSpec=0 ; iSpec < dustDensity->numSpec ; iSpec++){
      cumen = dustTemperature->cumulEner[iSpec][iz][iy][ix] / (dustDensity->densities[iSpec][iz][iy][ix]* grid->cellVolumes);
      dustTemperature->temperatures[iSpec][iz][iy][ix] = computeDusttempEnergyBd(emissivityDb, cumen, iSpec);
    }
  }

__device__ void doAbsorptionEvent(Photon* photon, Grid* grid, FrequenciesData* freqData, Stars* stars,
  DustDensity* dustDensity, DustOpacity* dustOpacity, EmissivityDatabase* emissivityDb, DustTemperature* dustTemperature){
    int ix = photon->gridPosition[0];
    int iy = photon->gridPosition[1];
    int iz = photon->gridPosition[2];
    divideAbsorvedEnergy(photon,stars,dustDensity, dustOpacity);
    addTemperatureDecoupled(photon, dustDensity, grid, emissivityDb, dustTemperature);
    for (int i=0 ; i<dustDensity->numSpec ; i++){
      photon->tempLocal[i] = dustTemperature->temperatures[i][iz][iy][ix];
    }
    photon->iFrequency = pickRandomFreqDb(emissivityDb, photon, dustDensity->numSpec, freqData->numFrequencies, photon->tempLocal, photon->enerPart);
    //photon->iFrequency = 55;
    //printf("newInu=%d\n",photon->iFrequency);
  }

__device__ double advanceToNextPositionTest(Photon* photon, Grid* grid, double cellWallsX[], double cellWallsY[], double cellWallsZ[]){
  //obtain orientations. It is 0 (left,down) or 1 (right, up)
  int ix = floor(photon->direction[0])+1.0;
  int iy = floor(photon->direction[1])+1.0;
  int iz = floor(photon->direction[2])+1.0;

  photon->orientations[0]=ix;
  photon->orientations[1]=iy;
  photon->orientations[2]=iz;

  //test shared memory
  //axis x
  photon->cellWalls[0] = cellWallsX[photon->gridPosition[0]+ix];
  //axis y
  photon->cellWalls[1] = cellWallsY[photon->gridPosition[1]+iy];
  //axis z
  photon->cellWalls[2] = cellWallsZ[photon->gridPosition[2]+iz];
  //printf("cellWallsX=%10.10lg\n",cellWallsX[photon->gridPosition[0]+ix]);


  //get 3 walls of grid position
  //getCellWalls(photon, grid, photon->gridPosition, photon->orientations);

  //distance to axis x
  photon->distances[0] = (photon->cellWalls[0] - photon->rayPosition[0]) / photon->direction[0];
  //distance to axis y
  photon->distances[1] = (photon->cellWalls[1] - photon->rayPosition[1]) / photon->direction[1];
  //distance to axis z
  photon->distances[2] = (photon->cellWalls[2] - photon->rayPosition[2]) / photon->direction[2];
  //printf("distances: %lf, %lf, %lf\n",distances[0],distances[1],distances[2]);

  //calculate min distance
  double tmp = fmin(photon->distances[0], photon->distances[1]);
  double minDistance = fmin(tmp, photon->distances[2]);
  //printf("minDistance: %lf\n", minDistance);

  //obtain minimun's axis
  //can be more than 1 (corners)
  int count = 0;
  int indexes[3] = {-1,-1,-1};
  for (int i=0 ; i<3 ; i++){
    if (photon->distances[i] == minDistance){
      indexes[count]=i;
      count++;
    }
  }
  //printf("minDistance, count: %lf, %d\n", minDistance, count);

  //update ray position
  photon->rayPosition[0] += minDistance*photon->direction[0];
  photon->rayPosition[1] += minDistance*photon->direction[1];
  photon->rayPosition[2] += minDistance*photon->direction[2];

  //avoid bug assign cellWall to ray position
  //update grid position with signs
  for (int i=0 ; i<count ; i++){
    photon->rayPosition[indexes[i]] = photon->cellWalls[indexes[i]];
    photon->gridPosition[indexes[i]] += signs[photon->orientations[indexes[i]]];
  }

  //is photon on the grid or outside?
  int nx = grid->nCoord[0];
  int ny = grid->nCoord[1];
  int nz = grid->nCoord[2];
  bool onX = (photon->gridPosition[0] >= 0) && (photon->gridPosition[0] < nx);
  bool onY = (photon->gridPosition[1] >= 0) && (photon->gridPosition[1] < ny);
  bool onZ = (photon->gridPosition[2] >= 0) && (photon->gridPosition[2] < nz);
  photon->onGrid = (onX && onY && onZ);
  return minDistance;

}

__device__ double advanceToNextPosition(Photon* photon, Grid* grid){
  //obtain orientations. It is 0 (left,down) or 1 (right, up)
  //int ix = floor(photon->direction[0])+1.0;
  //int iy = floor(photon->direction[1])+1.0;
  //int iz = floor(photon->direction[2])+1.0;

  //photon->orientations[0]=ix;
  //photon->orientations[1]=iy;
  //photon->orientations[2]=iz;


  //get 3 walls of grid position
  getCellWalls(photon, grid, photon->gridPosition, photon->orientations);

  //distance to axis x
  photon->distances[0] = (photon->cellWalls[0] - photon->rayPosition[0]) / photon->direction[0];
  //distance to axis y
  photon->distances[1] = (photon->cellWalls[1] - photon->rayPosition[1]) / photon->direction[1];
  //distance to axis z
  photon->distances[2] = (photon->cellWalls[2] - photon->rayPosition[2]) / photon->direction[2];
  //printf("distances: %lf, %lf, %lf\n",distances[0],distances[1],distances[2]);

  //calculate min distance
  double tmp = fmin(photon->distances[0], photon->distances[1]);
  double minDistance = fmin(tmp, photon->distances[2]);
  //printf("minDistance: %lf\n", minDistance);

  //obtain minimun's axis
  //can be more than 1 (corners)
  int count = 0;
  int indexes[3] = {-1,-1,-1};
  for (int i=0 ; i<3 ; i++){
    if (photon->distances[i] == minDistance){
      indexes[count]=i;
      count++;
    }
  }
  //printf("minDistance, count: %lf, %d\n", minDistance, count);

  //update ray position
  photon->rayPosition[0] += minDistance*photon->direction[0];
  photon->rayPosition[1] += minDistance*photon->direction[1];
  photon->rayPosition[2] += minDistance*photon->direction[2];

  //avoid bug assign cellWall to ray position
  //update grid position with signs
  for (int i=0 ; i<count ; i++){
    photon->rayPosition[indexes[i]] = photon->cellWalls[indexes[i]];
    photon->gridPosition[indexes[i]] += signs[photon->orientations[indexes[i]]];
  }

  //is photon on the grid or outside?
  photon->onGrid = photonIsOnGrid(photon, grid);
  return minDistance;
}

__device__ bool photonIsOnGrid(Photon* photon, Grid* grid){
  bool onX = (photon->gridPosition[0] >= 0) && (photon->gridPosition[0] < grid->nCoord[0]);
  bool onY = (photon->gridPosition[1] >= 0) && (photon->gridPosition[1] < grid->nCoord[1]);
  bool onZ = (photon->gridPosition[2] >= 0) && (photon->gridPosition[2] < grid->nCoord[2]);
  bool onGrid = (onX && onY && onZ);
  return onGrid;
}

__device__ void getTaupath(Photon* photon, hiprandState* state){
  float rn = hiprand_uniform(state);
  photon->taupathTotal = - log(1.0-rn);
  photon->taupathGone = 0.0;
}

__device__ void findNewFrequencyInu(Photon* photon, hiprandState* state, double* specCum, FrequenciesData* freqData){
  float rn = hiprand_uniform(state);
  int freq= (int)photon->iFrequency;
  huntDouble(specCum, freqData->numFrequencies+1, (double) rn, &freq);
  photon->iFrequency = (short)freq;
  //printf("rayInu = %d\n",rayInu);
  //return rayInu;
}

__device__ void checkUnitVector(float x, float y, float z){
    float module = sqrt(x*x + y*y + z*z);
    if (fabs(module - 1.0) > 1e-6){
        printf("%lf %lf %lf Error unity vector\n",x,y,z);
    }else{
      //printf("%2.8lg %2.8lg %2.8lg correct\n",x,y,z);
    }
}

__device__ void getRandomDirectionSimple(Photon* photon, hiprandState *state){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  float dirx=0, diry=0, dirz=0;
  float rnX;
  float rnY;
  float rnZ;
  //ol correctRandoms = false;
  bool equalZero = true;
  //bool equalOne = true;
  float tmp, module;


  while (equalZero){
    tmp = 2.0;
    while(tmp > 1){
      rnX=hiprand_uniform(&photon->state);
      rnY=hiprand_uniform(&photon->state);
      rnZ=hiprand_uniform(&photon->state);
      //correctRandoms = (rnX != 0.5) && (rnY != 0.5) && (rnZ != 0.5);

      dirx = 2*rnX-1;
      diry = 2*rnY-1;
      dirz = 2*rnZ-1;

      tmp = dirx*dirx + diry*diry + dirz*dirz;
      if (tmp < 0.0001){
        tmp = 2.0;
      }
    }
    //printf("direction =%lf,%lf,%lf\n",dirx,diry,dirz);
    //printf("direction =%lf,%lf,%lf\n",dirx,diry,dirz);
    equalZero = (dirx==0.0) || (diry==0.0)|| (dirz==0.0);
  }
    //printf("tmp %d= %f\n",tid,tmp);

  module = 1.0/sqrtf(tmp);
  dirx = dirx*module;
  diry = diry*module;
  dirz = dirz*module;

  //equalOne =  (dirx==1.0) || (diry==1.0)|| (dirz==1.0);
  //printf("direction %d=%lf,%lf,%lf\n",tid,photon->direction[0],photon->direction[1],photon->direction[2]);

  checkUnitVector(dirx,diry,dirz);


  //get orientations
  //obtain orientations. It is 0 (left,down) or 1 (right, up)
  int ix = floor(dirx)+1.0;
  int iy = floor(diry)+1.0;
  int iz = floor(dirz)+1.0;

  photon->orientations[0]=valuesOrientations[ix];
  photon->orientations[1]=valuesOrientations[iy];
  photon->orientations[2]=valuesOrientations[iz];

  photon->direction[0] = dirx;
  photon->direction[1] = diry;
  photon->direction[2] = dirz;

  //printf("direction=%lf,%lf,%lf\norientations %d, %d, %d\n",dirx,diry,dirz,photon->orientations[0],photon->orientations[1],photon->orientations[2]);
  return;
}

__device__ void radmcHenveyGreensteinDirection(Photon* photon, DustOpacity* dustOpacity, int iSpec){
  float newx,newy,newz;
  float temp, g2, sign;
  bool equalZero = true;
  //printf("dustOpacity->g[iSpec][photon->iFrequency]=%10.10lg\n",dustOpacity->g[iSpec][photon->iFrequency]);
  double g = dustOpacity->g[iSpec][photon->iFrequency];

  //get random numbers != 0.5 , and 2*random-1 != 0
  float rnX = 0.5;
  float rnY = 0.5;
  float rnZ = 0.5;

  while (rnX != 0.5){
    rnX=hiprand_uniform(&photon->state);
  }

  newx = 2*rnX-1;
  if (g > 0){
    //float rn = hiprand_uniform(&photon->state);
    g2 = g*g;
    temp = (1.0-g2)/(1.0 + g*newx);
    newx = (1.0+g2 - temp*temp)/(2.0*g);
    newx = fmaxf(newx,-1.0);
    newx = fminf(newx,1.0);
  }


  float l2 = 2.0;
  while(l2>1.0){
      rnY=hiprand_uniform(&photon->state);
      rnZ=hiprand_uniform(&photon->state);
    newy =2*rnY-1;
    newz =2*rnZ-1;
    l2 = newy*newy + newz*newz;
    l2 = (l2 < 0.0001 ? 2.0 : l2);
  }

  float linv = sqrtf((1.0-newx*newx)/l2);
  newy= newy*linv;
  newz= newz*linv;

  float oldx=photon->direction[0];
  float oldy=photon->direction[1];
  float oldz=photon->direction[2];

  //rotateVector
  float l = sqrtf(oldx*oldx+oldy*oldy);
  float vx = l*newx-oldz*newz;
  float vy=newy;
  float vz=oldz*newx+l*newz;
  newx = vx;
  newz = vz;
  if (l>1e-6){
    float dx=oldx/l;
    float dy=oldy/l;
    newx=dx*vx-dy*vy;
    newy=dy*vx+dx*vy;
  }
  checkUnitVector(newx,newy,newz);
  //get orientations
  //obtain orientations. It is 0 (left,down) or 1 (right, up)
  int ix = floorf(newx)+1.0;
  int iy = floorf(newy)+1.0;
  int iz = floorf(newz)+1.0;

  photon->orientations[0]=valuesOrientations[ix];
  photon->orientations[1]=valuesOrientations[iy];
  photon->orientations[2]=valuesOrientations[iz];
  //printf("floor: %d, %d, %d\n",ix,iy,iz);
  /*if (ix==2 || iy==2 ||iz==2 ){
    printf("actual: dirx=%lf diry=%lf dirz=%lf\nnew: dirx=%lf diry=%lf dirz=%lf\n",photon->direction[0],photon->direction[1],photon->direction[2],newx,newy,newz);
  }

  if (ix==2 || iy==2 ||iz==2 ){
    checkUnitVector(newx,newy,newz);
    //printf("newx=%2.8lg newy=%2.8lg newz=%2.8lg\n",newx,newy,newz);
  }*/
  photon->direction[0]=newx;
  photon->direction[1]=newy;
  photon->direction[2]=newz;
  return;

}

__device__ void getHenveyGreensteinDirection(Photon* photon, DustOpacity* dustOpacity, int iSpec){
  //printf("iSpec=%d, ifreq=%d\n",iSpec,photon->iFrequency);
  float newx,newy,newz;
  float temp, g2, sign;
  //printf("dustOpacity->g[iSpec][photon->iFrequency]=%10.10lg\n",dustOpacity->g[iSpec][photon->iFrequency]);
  double g = dustOpacity->g[iSpec][photon->iFrequency];
  float cosTheta = 2*hiprand_uniform(&photon->state)-1;
  if (g > 0){
    //float rn = hiprand_uniform(&photon->state);
    g2 = g*g;
    temp = (1.0-g2)/(1.0 + g*cosTheta);
    cosTheta = (1.0+g2 - temp*temp)/(2.0*g);
    cosTheta = fmaxf(cosTheta,-1.0);
    cosTheta = fminf(cosTheta,1.0);
  }


  float sinTheta = sqrtf(1-cosTheta*cosTheta);
  float psi = 2*PI*hiprand_uniform(&photon->state);
  float cosPsi = cosf(psi);
  float sinPsi = sqrtf(1-cosPsi*cosPsi);
  float sinTcosP = sinTheta*cosPsi;
  float sinTsinP = sinTheta*sinPsi;
  float last_ux = photon->direction[0];
  float last_uy = photon->direction[1];
  float last_uz = photon->direction[2];


  if (fabsf(last_uz) >= 0.99999  || fabsf(last_uy) >= 0.99999 || fabsf(last_ux) >= 0.99999){  //|| fabsf(last_uy) >= 0.99999 || fabsf(last_ux) >= 0.99999
    sign = ((last_uz) >= 0 ? 1 : -1);
    newx = sinTcosP;
    newy = sinTsinP;
    newz = cosTheta*sign;
  }else{
    temp = 1.0 / sqrtf(1 - last_uz * last_uz);
    newx = (sinTcosP * last_ux * last_uz - sinTsinP * last_uy) * temp + last_ux * cosTheta;
    newy = (sinTcosP * last_uy * last_uz + sinTsinP * last_ux) * temp + last_uy * cosTheta;
    newz = (-sinTcosP / temp) + last_uz * cosTheta;
  }
  checkUnitVector(newx,newy,newz);
  /*if (ix==2 || iy==2 ||iz==2 ){
    printf("actual: dirx=%lf diry=%lf dirz=%lf\nnew: dirx=%lf diry=%lf dirz=%lf\n",photon->direction[0],photon->direction[1],photon->direction[2],newx,newy,newz);
  }*/
  /*if (ix==2 || iy==2 ||iz==2 ){
    checkUnitVector(newx,newy,newz);
    //printf("newx=%2.8lg newy=%2.8lg newz=%2.8lg\n",newx,newy,newz);
  }*/
  photon->direction[0]=newx;
  photon->direction[1]=newy;
  photon->direction[2]=newz;
  return;
}
/*
__device__ void getHenveyGreensteinDirection(Photon* photon, DustOpacity* dustOpacity, double g){
  float newx,newy,newz;
  bool equalZero = true;
  bool equalOne = true;
  float rnx = hiprand_uniform(&photon->state);
  if (g != 0){
    float g2 = g*g;
    float tmp = (1.0-g2)/(1-g+2*g*rn);
    newx = (0.5*g)*(1.0+g2 - pow(tmp,2));
  }else{
    newx = 2*hiprand_uniform(&photon->state)-1;
  }

}*/

__device__ int findSpeciesToScattering(Photon* photon, int numSpec){
  int iSpec=0;
  photon->alphaCum[0] = 0.0;
  for (int i=0 ; i<numSpec ; i++){
    photon->alphaCum[i+1] = photon->alphaCum[i]+photon->alphaSSpec[i];
  }
  for (int i=0 ; i<numSpec ; i++){
    photon->alphaCum[i] = photon->alphaCum[i]+photon->alphaCum[numSpec];
  }
  photon->alphaCum[numSpec] = 1.0;
  float rn = hiprand_uniform(&photon->state);
  huntDouble(photon->alphaCum, numSpec+1, (double) rn, &iSpec);
  return iSpec;
}

__device__ void findStar(Photon* photon, Stars* d_stars, hiprandState* state){
  photon->iStar = 0;
  if (d_stars->numStars > 1){
    float rn = hiprand_uniform(state);
    int istar=(int)photon->iStar;
    //printf("star lumcum %10.10lg %10.10lg %10.10lg\n",d_stars->luminositiesCum[0],d_stars->luminositiesCum[1],d_stars->luminositiesCum[2]);
    //for (int i=0 ; i<d_stars->numStars+1 ; i++){
    //  printf("%10.10lg ",d_stars->luminositiesCum[i]);
    //}
  //  printf("\n");
    huntDouble(d_stars->luminositiesCum, d_stars->numStars+1, (double) rn, &istar);
    photon->iStar = (short) istar;
  }
}

__host__ void setUpPhoton(Photon* photon, int numSpec, int numFreq){
  photon->alphaCum = (double*)malloc(sizeof(double)*(numSpec+1));
  photon->alphaASpec = (double*)malloc(sizeof(double)*numSpec);
  photon->alphaSSpec = (double*)malloc(sizeof(double)*numSpec);
  photon->dbCumul = (float*)malloc(sizeof(float)*(numFreq+1));
  photon->enerCum = (double*)malloc(sizeof(double)*(numSpec+1));
  photon->enerPart = (double*)malloc(sizeof(double)*numSpec);
  photon->tempLocal = (double*)malloc(sizeof(double)*numSpec);
  photon->onGrid = true;
  /*photon->taupathGone = 0;
  photon->taupathTotal = 0;
  photon->iFrequency = 0;
  for (int i=0 ; i<numSpec ; i++){
    photon->alphaASpec[i] = 0;
    photon->alphaSSpec[i] = 0;
  }*/
}

__host__ void freePhoton(Photon* photon){
  free(photon->alphaASpec);
  free(photon->alphaSSpec);
  free(photon->dbCumul);
  free(photon->enerCum);
  free(photon->enerPart);
  free(photon->tempLocal);
}

__device__ void calculateOpacityCoefficients(double minorDistance, Photon* photon, DustDensity* dustDensity,
  DustOpacity* dustOpacity){
    int ix = photon->prevGridPosition[0];
    int iy = photon->prevGridPosition[1];
    int iz = photon->prevGridPosition[2];
    photon->opacCoeff.alphaATotal=0;
    photon->opacCoeff.alphaSTotal=0;
    for (int iSpec=0 ; iSpec<dustDensity->numSpec ; iSpec++){
      //printf("densities iSpec%d: %10.10lg\n",iSpec,dustDensity->densities[iSpec][iz][iy][ix]);
      //printf("kappaA iSpec%d: %10.10lg\n",iSpec,dustOpacity->kappaA[iSpec][photon->iFrequency]);
      photon->alphaASpec[iSpec] = dustDensity->densities[iSpec][iz][iy][ix]*dustOpacity->kappaA[iSpec][photon->iFrequency];
      photon->alphaSSpec[iSpec] = dustDensity->densities[iSpec][iz][iy][ix]*dustOpacity->kappaS[iSpec][photon->iFrequency];
      photon->opacCoeff.alphaATotal += photon->alphaASpec[iSpec];
      photon->opacCoeff.alphaSTotal += photon->alphaSSpec[iSpec];
      //printf("iSpec%d alphaATotal: %10.10lg\n",iSpec, photon->opacCoeff.alphaATotal);
    }
    photon->opacCoeff.alphaTotal = photon->opacCoeff.alphaATotal+photon->opacCoeff.alphaSTotal;
    photon->opacCoeff.albedo = photon->opacCoeff.alphaSTotal/photon->opacCoeff.alphaTotal;
    photon->opacCoeff.dtau = photon->opacCoeff.alphaTotal * minorDistance;
    //printf("dtau: %10.10lg\n",opacCoeff->dtau);
}

__device__ void inicializePositionPhoton(Photon* photon, Stars* d_stars,
  Grid* d_grid){
  //rayPosition
  photon->rayPosition[0] = d_stars->positions[photon->iStar][0];
  photon->rayPosition[1] = d_stars->positions[photon->iStar][1];
  photon->rayPosition[2] = d_stars->positions[photon->iStar][2];

  //gridPosition
  convertRayToGrid(photon, d_grid);
}

__device__ void walkNextEvent(Photon* d_photons, Stars* d_stars, DustDensity* d_dustDensity, DustOpacity* d_dustOpacity,
    Grid* d_grid, DustTemperature* d_dustTemperature){ //double cellWallsX[], double cellWallsY[], double cellWallsZ[]
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  double minorDistance, fraction, dum, addTmp;
  bool carryOn = true;
  //printf("in walkEvent\n");
  //printf("cellWallsX 0 = %10.10lg\n",cellWallsX[0] );
  //printf("cellWallsX 1 = %10.10lg\n",cellWallsX[1] );
  while (carryOn){

    d_photons[tid].prevRayPosition[0] = d_photons[tid].rayPosition[0];
    d_photons[tid].prevRayPosition[1] = d_photons[tid].rayPosition[1];
    d_photons[tid].prevRayPosition[2] = d_photons[tid].rayPosition[2];

    d_photons[tid].prevGridPosition[0] = d_photons[tid].gridPosition[0];
    d_photons[tid].prevGridPosition[1] = d_photons[tid].gridPosition[1];
    d_photons[tid].prevGridPosition[2] = d_photons[tid].gridPosition[2];
    //printf("start advanceToNextPosition\n");
    //minorDistance = advanceToNextPositionTest(&d_photons[tid], d_grid,cellWallsX,cellWallsY,cellWallsZ);
    minorDistance = advanceToNextPosition(&d_photons[tid], d_grid);
    //printf("newPosition: %d, %d, %d\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2]);
    //if (d_photons[tid].gridPosition[0]==0 && d_photons[tid].gridPosition[1]==0 && d_photons[tid].gridPosition[2]==0){
    //  printf("GridPosition: %d, %d, %d, onGrid = %d\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2], d_photons[tid].onGrid);
    //}

    calculateOpacityCoefficients(minorDistance, &d_photons[tid], d_dustDensity, d_dustOpacity);
    //printf("alphaATotal=%10.10lg\n", d_photons[tid].opacCoeff.alphaATotal);
    if (d_photons[tid].taupathGone + d_photons[tid].opacCoeff.dtau > d_photons[tid].taupathTotal){
      //printf("taupathGone=%lf, dtau=%lf, taupathTotal=%lf\n", d_photons[tid].taupathGone,d_photons[tid].opacCoeff.dtau,d_photons[tid].taupathTotal);
      //printf("\nReached end point!\n\n");
      fraction = (d_photons[tid].taupathTotal - d_photons[tid].taupathGone)/d_photons[tid].opacCoeff.dtau;
      //printf("fr=%lf\n", fraction)
      //update ray position

      d_photons[tid].rayPosition[0] = d_photons[tid].prevRayPosition[0] + fraction * (d_photons[tid].rayPosition[0] - d_photons[tid].prevRayPosition[0]);
      d_photons[tid].rayPosition[1] = d_photons[tid].prevRayPosition[1] + fraction * (d_photons[tid].rayPosition[1] - d_photons[tid].prevRayPosition[1]);
      d_photons[tid].rayPosition[2] = d_photons[tid].prevRayPosition[2] + fraction * (d_photons[tid].rayPosition[2] - d_photons[tid].prevRayPosition[2]);
      //update grid position
      d_photons[tid].gridPosition[0] = d_photons[tid].prevGridPosition[0];
      d_photons[tid].gridPosition[1] = d_photons[tid].prevGridPosition[1];
      d_photons[tid].gridPosition[2] = d_photons[tid].prevGridPosition[2];
      //index = getIndexFromIxyz(d_grid, photon->gridPosition);
      dum = (1.0-d_photons[tid].opacCoeff.albedo) * (d_photons[tid].taupathTotal-d_photons[tid].taupathGone) * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
      //printf("dum=%10.10lg\n",dum);
      for (int i=0 ; i<d_dustDensity->numSpec ; i++){
        //printf("numSpec=%d\n",d_dustDensity->numSpec);
        addTmp = dum*d_photons[tid].alphaASpec[i];
        //d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]] += addTmp;

        atomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]), addTmp);
        /*if (d_photons[tid].gridPosition[0]==0 && d_photons[tid].gridPosition[1]==0 && d_photons[tid].gridPosition[2]==0){
          printf("cumulEner %d, %d, %d = %10.10lg\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2],
          d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
        }*/
        //d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]] += addTmp;
        //printf("cumulEner=%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]);
      }
      carryOn = false;
    }else{
      dum = (1.0-d_photons[tid].opacCoeff.albedo) * d_photons[tid].opacCoeff.dtau * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
      for (int i=0 ; i<d_dustDensity->numSpec ; i++){
        addTmp = dum*d_photons[tid].alphaASpec[i];
        //d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]] += addTmp;
        atomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]), addTmp);
        //if (d_photons[tid].prevGridPosition[0]==0 && d_photons[tid].prevGridPosition[1]==0 && d_photons[tid].prevGridPosition[2]==0){
        //  printf("cumulEner %d, %d, %d = %10.10lg\n",d_photons[tid].prevGridPosition[0],d_photons[tid].prevGridPosition[1],d_photons[tid].prevGridPosition[2],
        //  d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
        //}
        //d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]] += addTmp;
        //printf("cumulEner =%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
      }

      d_photons[tid].taupathGone += d_photons[tid].opacCoeff.dtau;
      carryOn = d_photons[tid].onGrid;
    }
  }
  //printf("FinalRayPosition: %lf, %lf, %lf\n",d_photons[tid].rayPosition[0],d_photons[tid].rayPosition[1],d_photons[tid].rayPosition[2]);
  //printf("FinalGridPosition: %d, %d, %d\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2]);
  float rn = hiprand_uniform(&d_photons[tid].state);
  //printf("rn=%lf\n",rn);
  d_photons[tid].isScattering = rn < d_photons[tid].opacCoeff.albedo;
  //printf("end walkEvent\n");
}
/*
__device__ void walkNextEvent(Photon* d_photons, Stars* d_stars, DustDensity* d_dustDensity, DustOpacity* d_dustOpacity,
    Grid* d_grid, DustTemperature* d_dustTemperature){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  double minorDistance, fraction, dum, addTmp;
  bool carryOn = true;
  //printf("cellWallsX 0 = %10.10lg\n",cellWallsX[0] );
  //printf("cellWallsX 1 = %10.10lg\n",cellWallsX[1] );
  while (carryOn){

    d_photons[tid].prevRayPosition[0] = d_photons[tid].rayPosition[0];
    d_photons[tid].prevRayPosition[1] = d_photons[tid].rayPosition[1];
    d_photons[tid].prevRayPosition[2] = d_photons[tid].rayPosition[2];

    d_photons[tid].prevGridPosition[0] = d_photons[tid].gridPosition[0];
    d_photons[tid].prevGridPosition[1] = d_photons[tid].gridPosition[1];
    d_photons[tid].prevGridPosition[2] = d_photons[tid].gridPosition[2];
    //printf("start advanceToNextPosition\n");
    //minorDistance = advanceToNextPositionTest(&d_photons[tid], d_grid,cellWallsX,cellWallsY,cellWallsZ);
    minorDistance = advanceToNextPosition(&d_photons[tid], d_grid);
    if (d_photons[tid].gridPosition[0]==0 && d_photons[tid].gridPosition[1]==0 && d_photons[tid].gridPosition[2]==0){
      printf("GridPosition: %d, %d, %d, onGrid = %d\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2], d_photons[tid].onGrid);
    }

    if (d_photons[tid].onGrid){
      calculateOpacityCoefficients(minorDistance, &d_photons[tid], d_dustDensity, d_dustOpacity);

      //printf("alphaSTotal=%10.10lg\n", opacCoeff->alphaSTotal);

      if (d_photons[tid].taupathGone + d_photons[tid].opacCoeff.dtau > d_photons[tid].taupathTotal){
        //printf("taupathGone=%lf, dtau=%lf, taupathTotal=%lf\n", d_photons[tid].taupathGone,d_photons[tid].opacCoeff.dtau,d_photons[tid].taupathTotal);
        //printf("\nReached end point!\n\n");
        fraction = (d_photons[tid].taupathTotal - d_photons[tid].taupathGone)/d_photons[tid].opacCoeff.dtau;
        //printf("fr=%lf\n", fraction)
        //update ray position

        d_photons[tid].rayPosition[0] = d_photons[tid].prevRayPosition[0] + fraction * (d_photons[tid].rayPosition[0] - d_photons[tid].prevRayPosition[0]);
        d_photons[tid].rayPosition[1] = d_photons[tid].prevRayPosition[1] + fraction * (d_photons[tid].rayPosition[1] - d_photons[tid].prevRayPosition[1]);
        d_photons[tid].rayPosition[2] = d_photons[tid].prevRayPosition[2] + fraction * (d_photons[tid].rayPosition[2] - d_photons[tid].prevRayPosition[2]);
        //update grid position
        d_photons[tid].gridPosition[0] = d_photons[tid].prevGridPosition[0];
        d_photons[tid].gridPosition[1] = d_photons[tid].prevGridPosition[1];
        d_photons[tid].gridPosition[2] = d_photons[tid].prevGridPosition[2];
        //index = getIndexFromIxyz(d_grid, photon->gridPosition);
        dum = (1.0-d_photons[tid].opacCoeff.albedo) * (d_photons[tid].taupathTotal-d_photons[tid].taupathGone) * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
        //printf("dum=%10.10lg\n",dum);
        for (int i=0 ; i<d_dustDensity->numSpec ; i++){
          addTmp = dum*d_photons[tid].alphaASpec[i];
          //d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]] += addTmp;

          atomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]), addTmp);
          if (d_photons[tid].gridPosition[0]==0 && d_photons[tid].gridPosition[1]==0 && d_photons[tid].gridPosition[2]==0){
            printf("cumulEner %d, %d, %d = %10.10lg\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2],
            d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
          }
          //d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]] += addTmp;
          //printf("cumulEner=%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]);
        }
        carryOn = false;
      }else{
        dum = (1.0-d_photons[tid].opacCoeff.albedo) * d_photons[tid].opacCoeff.dtau * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
        for (int i=0 ; i<d_dustDensity->numSpec ; i++){
          addTmp = dum*d_photons[tid].alphaASpec[i];
          //d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]] += addTmp;
          atomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]), addTmp);
          if (d_photons[tid].prevGridPosition[0]==0 && d_photons[tid].prevGridPosition[1]==0 && d_photons[tid].prevGridPosition[2]==0){
            printf("cumulEner %d, %d, %d = %10.10lg\n",d_photons[tid].prevGridPosition[0],d_photons[tid].prevGridPosition[1],d_photons[tid].prevGridPosition[2],
            d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
          }
          //d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]] += addTmp;
          //printf("cumulEner =%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
        }

        d_photons[tid].taupathGone += d_photons[tid].opacCoeff.dtau;
      }
      //if photon is outside of grid
    }else{
      carryOn = false;
    }
          //printf("Ongrid? = %d\n",d_photons[tid].onGrid);
  }
  //printf("FinalRayPosition: %lf, %lf, %lf\n",d_photons[tid].rayPosition[0],d_photons[tid].rayPosition[1],d_photons[tid].rayPosition[2]);
  //printf("FinalGridPosition: %d, %d, %d\n",d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2]);
  float rn = hiprand_uniform(&d_photons[tid].state);
  //printf("rn=%lf\n",rn);
  d_photons[tid].isScattering = rn < d_photons[tid].opacCoeff.albedo;
  //printf("end walkEvent\n");
}*/

__global__ void kernelWalkNextEvent(Photon* d_photons, Stars* d_stars, DustDensity* d_dustDensity, DustOpacity* d_dustOpacity,
  Grid* d_grid, DustTemperature* d_dustTemperature){
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (d_photons[tid].onGrid){
      //printf("startGridPosition %d: %d, %d, %d, ongrid=%d\n",tid,d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2],d_photons[tid].onGrid);

      double minorDistance, fraction, dum, addTmp;
      bool carryOn = true;

      while (carryOn){
        //printf("\n");
        d_photons[tid].prevRayPosition[0] = d_photons[tid].rayPosition[0];
        d_photons[tid].prevRayPosition[1] = d_photons[tid].rayPosition[1];
        d_photons[tid].prevRayPosition[2] = d_photons[tid].rayPosition[2];

        d_photons[tid].prevGridPosition[0] = d_photons[tid].gridPosition[0];
        d_photons[tid].prevGridPosition[1] = d_photons[tid].gridPosition[1];
        d_photons[tid].prevGridPosition[2] = d_photons[tid].gridPosition[2];

        minorDistance = advanceToNextPosition(&d_photons[tid], d_grid);
        if (d_photons[tid].onGrid){
          calculateOpacityCoefficients(minorDistance, &d_photons[tid], d_dustDensity, d_dustOpacity);

          //printf("alphaSTotal=%10.10lg\n", opacCoeff->alphaSTotal);

          if (d_photons[tid].taupathGone + d_photons[tid].opacCoeff.dtau > d_photons[tid].taupathTotal){
            //printf("taupathGone=%lf, dtau=%lf, taupathTotal=%lf\n", d_photons[tid].taupathGone,d_photons[tid].opacCoeff.dtau,d_photons[tid].taupathTotal);
            //printf("\nReached end point!\n\n");
            fraction = (d_photons[tid].taupathTotal - d_photons[tid].taupathGone)/d_photons[tid].opacCoeff.dtau;
            //printf("fr=%lf\n", fraction)
            //update ray position

            d_photons[tid].rayPosition[0] = d_photons[tid].prevRayPosition[0] + fraction * (d_photons[tid].rayPosition[0] - d_photons[tid].prevRayPosition[0]);
            d_photons[tid].rayPosition[1] = d_photons[tid].prevRayPosition[1] + fraction * (d_photons[tid].rayPosition[1] - d_photons[tid].prevRayPosition[1]);
            d_photons[tid].rayPosition[2] = d_photons[tid].prevRayPosition[2] + fraction * (d_photons[tid].rayPosition[2] - d_photons[tid].prevRayPosition[2]);
            //update grid position
            d_photons[tid].gridPosition[0] = d_photons[tid].prevGridPosition[0];
            d_photons[tid].gridPosition[1] = d_photons[tid].prevGridPosition[1];
            d_photons[tid].gridPosition[2] = d_photons[tid].prevGridPosition[2];
            //index = getIndexFromIxyz(d_grid, photon->gridPosition);
            dum = (1-d_photons[tid].opacCoeff.albedo) * (d_photons[tid].taupathTotal-d_photons[tid].taupathGone) * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
            //printf("dum=%10.10lg\n",dum);
            for (int i=0 ; i<d_dustDensity->numSpec ; i++){
              printf("numSpec=%d\n",d_dustDensity->numSpec);
              addTmp = dum*d_photons[tid].alphaASpec[i];
              doubleAtomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]), addTmp);
              //printf("cumulEner=%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].gridPosition[2]][d_photons[tid].gridPosition[1]][d_photons[tid].gridPosition[0]]);
            }
            carryOn = false;
          }else{
            //index = getIndexFromIxyz(d_grid, photon->prevGridPosition);
            //printf("dum=%10.10lg\n",dum);
            dum = (1-d_photons[tid].opacCoeff.albedo) * d_photons[tid].opacCoeff.dtau * d_stars->energies[d_photons[tid].iStar] / d_photons[tid].opacCoeff.alphaATotal;
            for (int i=0 ; i<d_dustDensity->numSpec ; i++){
              addTmp = dum*d_photons[tid].alphaASpec[i];
              doubleAtomicAdd(&(d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]), addTmp);
              //printf("cumulEner =%10.10lg\n",d_dustTemperature->cumulEner[i][d_photons[tid].prevGridPosition[2]][d_photons[tid].prevGridPosition[1]][d_photons[tid].prevGridPosition[0]]);
            }
            d_photons[tid].taupathGone += d_photons[tid].opacCoeff.dtau;
          }
          //if photon is outside of grid
        }else{
          carryOn = false;
        }
              //printf("Ongrid? = %d\n",d_photons[tid].onGrid);
      }
      //printf("FinalGridPosition %d: %d, %d, %d, ongrid=%d\n",tid,d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2],d_photons[tid].onGrid);
      //printf("FinalRayPosition %d: %lf, %lf, %lf\n",tid, d_photons[tid].rayPosition[0],d_photons[tid].rayPosition[1],d_photons[tid].rayPosition[2]);
      float rn = hiprand_uniform(&d_photons[tid].state);
      //printf("rn=%lf\n",rn);
      d_photons[tid].isScattering = rn < d_photons[tid].opacCoeff.albedo;
    }
}

__global__ void inicializeInicialPhoton(Photon* d_photons, FrequenciesData* d_freqData, Stars* d_stars, Grid* d_grid,
  DustDensity* d_dustDensity, DustOpacity* d_dustOpacity){
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    //hiprandState state;
    hiprand_init((unsigned long long)clock()*tid, tid, 0, &d_photons[tid].state);
    //photons[tid] = setUpPhoton(d_dustDensity->numSpec, d_freqData->numFrequencies);
    findStar(&d_photons[tid],d_stars,&d_photons[tid].state);
    //printf("iStar=%d\n",d_photons[tid].iStar);
    inicializePositionPhoton(&d_photons[tid], d_stars, d_grid);
    //printf("GridPosition %d: %d, %d, %d\n",tid,d_photons[tid].gridPosition[0],d_photons[tid].gridPosition[1],d_photons[tid].gridPosition[2],d_photons[tid].onGrid);
    //printf("RayPosition %d: %lf, %lf, %lf\n",tid, d_photons[tid].rayPosition[0],d_photons[tid].rayPosition[1],d_photons[tid].rayPosition[2]);
    getRandomDirectionSimple(&d_photons[tid], &d_photons[tid].state);
    findNewFrequencyInu(&d_photons[tid],&d_photons[tid].state, d_stars->specCum[d_photons[tid].iStar], d_freqData);
    getTaupath(&d_photons[tid], &d_photons[tid].state);
    d_photons[tid].onGrid = true;
    //printf("iFrequency%d, iStar%d, taupath%lf, direction:%lf,%lf,%lf\n",
    //d_photons[tid].iFrequency, d_photons[tid].iStar, d_photons[tid].taupathTotal,
    //d_photons[tid].direction[0],d_photons[tid].direction[1],d_photons[tid].direction[2]);
}
/*
__global__ void restartValuesPhoton(Photon* d_photons){

}*/


__host__ Photon* allocatePhotons(int numPhotons, int numSpec, int numFreq){
  Photon* photons = (Photon*)malloc(sizeof(Photon)*numPhotons);
  for (int i=0 ; i< numPhotons ; i++){
    setUpPhoton(&photons[i], numSpec, numFreq);
  }
  return photons;
}

__host__ void deallocatePhotons(Photon* photons, int numPhotons){
  for (int i=0 ; i< numPhotons ; i++){
    freePhoton(&photons[i]);
  }
  free(photons);
}

__host__ Photon* photonsTransferToDevice(Photon* h_photons, int numPhotons, int numSpec, int numFreq){
  printf("Transfer photons to device...\n");
  Photon* d_photons;
  hipMalloc((void**)&(d_photons), sizeof(Photon)*numPhotons );
  Photon* photons = (Photon*)malloc(sizeof(Photon)*numPhotons);
  for (int i=0 ; i<numPhotons ; i++){
    hipMalloc((void**) &photons[i].alphaCum, sizeof(double)*(numSpec+1));
    hipMalloc((void**) &photons[i].alphaASpec, sizeof(double)*numSpec);
    hipMalloc((void**) &photons[i].alphaSSpec, sizeof(double)*numSpec);
    hipMalloc((void**) &photons[i].enerPart, sizeof(double)*numSpec);
    hipMalloc((void**) &photons[i].tempLocal, sizeof(double)*numSpec);
    hipMalloc((void**) &photons[i].enerCum, sizeof(double)*(numSpec+1));
    hipMalloc((void**) &photons[i].dbCumul, sizeof(float)*(numFreq+1));
  }
  hipMemcpy(d_photons, photons,sizeof(Photon)*numPhotons,hipMemcpyHostToDevice);
  free(photons);
  return d_photons;
}
/*
__global__ void printfPhotons(Photon* d_photons){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  d_photons[tid].opacCoeff.dtau = tid;
  printf("d_photons[tid].opacCoeff.dtau %lf\n",d_photons[tid].opacCoeff.dtau);
  //printf("Hola\n");
}*/

__global__ void launchPhotons(Photon* d_photons, FrequenciesData* d_freqData, Stars* d_stars, Grid* d_grid,
  DustDensity* d_dustDensity, DustOpacity* d_dustOpacity,
  EmissivityDatabase* d_emissivityDb, DustTemperature* d_dustTemperature, SimulationParameters* d_params){
  //__shared__ double cellWallsX[257];
  //__shared__ double cellWallsY[257];
  //__shared__ double cellWallsZ[257];
  //printf("scatteringMode = %d\n",d_params->scatteringMode);
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
/*
  if (threadIdx.x < 63){
    for (int i=0 ; i<4 ; i++){
      int index = threadIdx.x*4+i;
      cellWallsX[index] = d_grid->cellWallsX[index];
      cellWallsY[index] = d_grid->cellWallsY[index];
      cellWallsZ[index] = d_grid->cellWallsZ[index];
      //printf("cellWallsX %d = %10.10lg\n",index,cellWallsX[index] );
    }
  }
  if (threadIdx.x == 63){
    for (int i=0 ; i<6 ; i++){
      int index = threadIdx.x*4+i;
      cellWallsX[index] = d_grid->cellWallsX[index];
      cellWallsY[index] = d_grid->cellWallsY[index];
      cellWallsZ[index] = d_grid->cellWallsZ[index];
    }
  }
  __syncthreads();*/
  //printf("hola\n");
  walkNextEvent(d_photons, d_stars, d_dustDensity, d_dustOpacity, d_grid, d_dustTemperature); // cellWallsX, cellWallsY, cellWallsZ
  //walkNextEvent(d_photons, d_stars, d_dustDensity, d_dustOpacity, d_grid, d_dustTemperature);
  while(d_photons[tid].onGrid){

    if (d_photons[tid].isScattering){
      //Do Scattering event
      if (d_params->scatteringMode == 1){
        //random direction
        //printf("scaterring\n");
        //printf("isotropic\n");
        getRandomDirectionSimple(&d_photons[tid], &d_photons[tid].state);
      }else if (d_params->scatteringMode == 2){
        //printf("anisotropic\n");

        //do henyey_greenstein_direction
        int iSpec = findSpeciesToScattering(&d_photons[tid], d_dustDensity->numSpec);
        //printf("dustOpacity->g[iSpec][photon->iFrequency] %10.10lg\n",d_dustOpacity->g[0][10]);
        radmcHenveyGreensteinDirection(&d_photons[tid], d_dustOpacity, iSpec);
        //getHenveyGreensteinDirection(&d_photons[tid], d_dustOpacity, iSpec);
      }

    }else{
      //printf("absoprtion\n");

      doAbsorptionEvent(&d_photons[tid], d_grid, d_freqData, d_stars, d_dustDensity,d_dustOpacity, d_emissivityDb, d_dustTemperature);
      //printf("newFreq = %d\n",d_photons[tid].iFrequency);
      //printf("start getRandomDirectionSimple\n");
      getRandomDirectionSimple(&d_photons[tid], &d_photons[tid].state);
      //printf("end getRandomDirectionSimple\n");
    }
    //printf("start getTaupath\n");
    getTaupath(&d_photons[tid], &d_photons[tid].state);
    //printf("start walkNextEvent, onGrid=%d\n",d_photons[tid].onGrid);
    walkNextEvent(d_photons, d_stars, d_dustDensity, d_dustOpacity, d_grid, d_dustTemperature);//,cellWallsX, cellWallsY, cellWallsZ
    //walkNextEvent(d_photons, d_stars, d_dustDensity, d_dustOpacity, d_grid, d_dustTemperature);
    //printf("onGrid? %d\n",d_photons[tid].onGrid);
  }
}
